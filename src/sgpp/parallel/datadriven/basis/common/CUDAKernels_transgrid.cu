#include "hip/hip_runtime.h"
/******************************************************************************
* Copyright (C) 2011 Technische Universitaet Muenchen                         *
* This file is part of the SG++ project. For conditions of distribution and   *
* use, please see the copyright notice at http://www5.in.tum.de/SGpp          *
******************************************************************************/
// @author Alexander Heinecke (Alexander.Heinecke@mytum.de)

#include "base/tools/SGppStopwatch.hpp"

#include "parallel/datadriven/basis/common/CUDAKernels.hpp"

// including CUDA
#include <hip/hip_runtime.h>

#include <iostream>

float* gpu_grid_level_sp;
float* gpu_grid_index_sp;
float* gpu_dataset_sp;
float* gpu_alpha_sp;
float* gpu_datavec_sp;
float* host_alpha_sp;
float* host_datavec_sp;
float* host_grid_level_sp;
float* host_grid_index_sp;
size_t gpu_full_storageSize;


__global__ void multTransSP_CUDA(float* ptrSource,
                                 float* ptrData,
                                 float* ptrLevel,
                                 float* ptrIndex,
                                 float* ptrResult,
                                 unsigned int sourceSize,
                                 unsigned int storageSize,
                                 unsigned int dims,
                                 unsigned int offset) {
  int globalIdx = (blockIdx.x * blockDim.x) + threadIdx.x + offset;
  int localIdx = threadIdx.x;
  float eval, index_calc, abs, last, localSupport, curSupport;
  float myResult = 0.0f;
  float locLevel[5];
  float locIndex[5];

  __shared__ float locData[320];
  __shared__ float locSource[64];

#pragma unroll 5

  for (unsigned int d = 0; d < dims; d++) {
    locLevel[d] = ptrLevel[globalIdx + (storageSize * d)];
    locIndex[d] = ptrIndex[globalIdx + (storageSize * d)];
  }

  // Iterate over all instances of the dataset
  for (unsigned int i = 0; i < sourceSize; i += 64) {
#pragma unroll 5

    for (unsigned int d = 0; d < dims; d++) {
      locData[(d*64)+localIdx] = ptrData[(d*sourceSize)+(i+localIdx)];
    }

    locSource[localIdx] = ptrSource[i + localIdx];

    // Wait until all data is in shared memory
    __syncthreads();

    for (unsigned int k = 0; k < 64; k++) {
      curSupport = locSource[k];

#pragma unroll 5

      for (unsigned int d = 0; d < dims; d++) {
        eval = locLevel[d] * locData[(d*64) + k];
        index_calc = eval - locIndex[d];
        abs = fabsf(index_calc);
        last = 1.0f - abs;
        localSupport = fmaxf(last, 0.0f);
        curSupport *= localSupport;
      }

      myResult += curSupport;
    }

    // Wait for calculation to finish (before new data can be loaded into shared memory)
    __syncthreads();
  }

  ptrResult[globalIdx] = myResult;
}

__global__ void multTransSP_CUDA_5d(float* ptrSource,
                                    float* ptrData,
                                    float* ptrLevel,
                                    float* ptrIndex,
                                    float* ptrResult,
                                    unsigned int sourceSize,
                                    unsigned int storageSize,
                                    unsigned int offset) {
  int globalIdx = (blockIdx.x * blockDim.x) + threadIdx.x + offset;
  int localIdx = threadIdx.x;
  float eval, index_calc, abs, last, localSupport, curSupport;
  float myResult = 0.0f;
  float locLevel_0;
  float locIndex_0;
  float locLevel_1;
  float locIndex_1;
  float locLevel_2;
  float locIndex_2;
  float locLevel_3;
  float locIndex_3;
  float locLevel_4;
  float locIndex_4;

  __shared__ float locData[320];
  __shared__ float locSource[64];

  locLevel_0 = ptrLevel[globalIdx + (storageSize * 0)];
  locIndex_0 = ptrIndex[globalIdx + (storageSize * 0)];
  locLevel_1 = ptrLevel[globalIdx + (storageSize * 1)];
  locIndex_1 = ptrIndex[globalIdx + (storageSize * 1)];
  locLevel_2 = ptrLevel[globalIdx + (storageSize * 2)];
  locIndex_2 = ptrIndex[globalIdx + (storageSize * 2)];
  locLevel_3 = ptrLevel[globalIdx + (storageSize * 3)];
  locIndex_3 = ptrIndex[globalIdx + (storageSize * 3)];
  locLevel_4 = ptrLevel[globalIdx + (storageSize * 4)];
  locIndex_4 = ptrIndex[globalIdx + (storageSize * 4)];

  // Iterate over all instances of the dataset
  for (unsigned int i = 0; i < sourceSize; i += 64) {
    locData[(0*64)+localIdx] = ptrData[(i + localIdx) + (0*sourceSize)];
    locData[(1*64)+localIdx] = ptrData[(i + localIdx) + (1*sourceSize)];
    locData[(2*64)+localIdx] = ptrData[(i + localIdx) + (2*sourceSize)];
    locData[(3*64)+localIdx] = ptrData[(i + localIdx) + (3*sourceSize)];
    locData[(4*64)+localIdx] = ptrData[(i + localIdx) + (4*sourceSize)];

    locSource[localIdx] = ptrSource[i + localIdx];

    // Wait until all data is in shared memory
    __syncthreads();

    for (unsigned int k = 0; k < 64; k++) {
      curSupport = locSource[k];

      eval = locLevel_0 * locData[(0*64)+k];
      index_calc = eval - locIndex_0;
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel_1 * locData[(1*64)+k];
      index_calc = eval - locIndex_1;
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel_2 * locData[(2*64)+k];
      index_calc = eval - locIndex_2;
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel_3 * locData[(3*64)+k];
      index_calc = eval - locIndex_3;
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel_4 * locData[(4*64)+k];
      index_calc = eval - locIndex_4;
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      myResult += curSupport;
    }

    // Wait for calculation to finish (before new data can be loaded into shared memory)
    __syncthreads();
  }

  ptrResult[globalIdx] = myResult;
}

__global__ void multSP_CUDA(float* ptrAlpha,
                            float* ptrData,
                            float* ptrLevel,
                            float* ptrIndex,
                            float* ptrResult,
                            unsigned int fastStorageSize,
                            unsigned int storageSize,
                            unsigned int dims,
                            unsigned int datasize,
                            unsigned int offset) {
  int globalIdx = (blockIdx.x * blockDim.x) + threadIdx.x + offset;
  int localIdx = threadIdx.x;
  float eval, index_calc, abs, last, localSupport, curSupport;
  float myResult = 0.0f;
  float locData[5];

  __shared__ float locLevel[320];
  __shared__ float locIndex[320];
  __shared__ float locAlpha[64];

#pragma unroll 5

  for (unsigned int d = 0; d < dims; d++) {
    locData[d] = ptrData[(d*datasize)+globalIdx];
  }

  // Iterate over all grid points (fast ones, with cache)
  for (unsigned int j = 0; j < fastStorageSize; j += 64) {
#pragma unroll 5

    for (unsigned int d = 0; d < dims; d++) {
      locLevel[localIdx + (64 * d)] = ptrLevel[(j + localIdx) + (storageSize * d)];
      locIndex[localIdx + (64 * d)] = ptrIndex[(j + localIdx) + (storageSize * d)];
    }

    locAlpha[localIdx] = ptrAlpha[j + localIdx];

    // Wait until all needed data is in shared memory
    __syncthreads();

    for (unsigned int k = 0; k < 64; k++) {
      curSupport = locAlpha[k];

#pragma unroll 5

      for (unsigned int d = 0; d < dims; d++) {
        eval = locLevel[k+(64*d)] * locData[d];
        index_calc = eval - locIndex[k+(64*d)];
        abs = fabsf(index_calc);
        last = 1.0f - abs;
        localSupport = fmaxf(last, 0.0f);
        curSupport *= localSupport;
      }

      myResult += curSupport;
    }

    // wait until calculations have finished (avoid overwriting shared memory)
    __syncthreads();
  }

  // Iterate over all grid points (slow ones, without cache)
  for (unsigned int m = fastStorageSize; m < storageSize; m++) {
    curSupport = ptrAlpha[m];

    for (unsigned int d = 0; d < dims; d++) {
      eval = ptrLevel[m+(storageSize*d)] * locData[d];
      index_calc = eval - ptrIndex[m+(storageSize*d)];
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;
    }

    myResult += curSupport;
  }

  ptrResult[globalIdx] = myResult;
}

__global__ void multSP_CUDA_5d(float* ptrAlpha,
                               float* ptrData,
                               float* ptrLevel,
                               float* ptrIndex,
                               float* ptrResult,
                               unsigned int fastStorageSize,
                               unsigned int storageSize,
                               unsigned int datasize,
                               unsigned int offset) {
  int globalIdx = (blockIdx.x * blockDim.x) + threadIdx.x + offset;
  int localIdx = threadIdx.x;
  float eval, index_calc, abs, last, localSupport, curSupport;
  float myResult = 0.0f;
  float locData_0;
  float locData_1;
  float locData_2;
  float locData_3;
  float locData_4;

  __shared__ float locLevel[320];
  __shared__ float locIndex[320];
  __shared__ float locAlpha[64];

  locData_0 = ptrData[(0*datasize)+globalIdx];
  locData_1 = ptrData[(1*datasize)+globalIdx];
  locData_2 = ptrData[(2*datasize)+globalIdx];
  locData_3 = ptrData[(3*datasize)+globalIdx];
  locData_4 = ptrData[(4*datasize)+globalIdx];

  // Iterate over all grid points (fast ones, with cache)
  for (unsigned int j = 0; j < fastStorageSize; j += 64) {
    locLevel[localIdx + (64*0)] = ptrLevel[(j + localIdx) + (storageSize*0)];
    locIndex[localIdx + (64*0)] = ptrIndex[(j + localIdx) + (storageSize*0)];
    locLevel[localIdx + (64*1)] = ptrLevel[(j + localIdx) + (storageSize*1)];
    locIndex[localIdx + (64*1)] = ptrIndex[(j + localIdx) + (storageSize*1)];
    locLevel[localIdx + (64*2)] = ptrLevel[(j + localIdx) + (storageSize*2)];
    locIndex[localIdx + (64*2)] = ptrIndex[(j + localIdx) + (storageSize*2)];
    locLevel[localIdx + (64*3)] = ptrLevel[(j + localIdx) + (storageSize*3)];
    locIndex[localIdx + (64*3)] = ptrIndex[(j + localIdx) + (storageSize*3)];
    locLevel[localIdx + (64*4)] = ptrLevel[(j + localIdx) + (storageSize*4)];
    locIndex[localIdx + (64*4)] = ptrIndex[(j + localIdx) + (storageSize*4)];

    locAlpha[localIdx] = ptrAlpha[j + localIdx];

    // Wait until all needed data is in shared memory
    __syncthreads();

    for (unsigned int k = 0; k < 64; k++) {
      curSupport = locAlpha[k];

      eval = locLevel[k + (0*64)] * locData_0;
      index_calc = eval - locIndex[k + (0*64)];
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel[k + (1*64)] * locData_1;
      index_calc = eval - locIndex[k + (1*64)];
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel[k + (2*64)] * locData_2;
      index_calc = eval - locIndex[k + (2*64)];
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel[k + (3*64)] * locData_3;
      index_calc = eval - locIndex[k + (3*64)];
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      eval = locLevel[k + (4*64)] * locData_4;
      index_calc = eval - locIndex[k + (4*64)];
      abs = fabsf(index_calc);
      last = 1.0f - abs;
      localSupport = fmaxf(last, 0.0f);
      curSupport *= localSupport;

      myResult += curSupport;
    }

    // wait until calculations have finished (avoid overwriting shared memory)
    __syncthreads();
  }

  // Iterate over all grid points (slow ones, without cache)
  for (unsigned int m = fastStorageSize; m < storageSize; m++) {
    curSupport = ptrAlpha[m];

    eval = ptrLevel[m + (0*storageSize)] * locData_0;
    index_calc = eval - ptrIndex[m + (0*storageSize)];
    abs = fabsf(index_calc);
    last = 1.0f - abs;
    localSupport = fmaxf(last, 0.0f);
    curSupport *= localSupport;

    eval = ptrLevel[m + (1*storageSize)] * locData_1;
    index_calc = eval - ptrIndex[m + (1*storageSize)];
    abs = fabsf(index_calc);
    last = 1.0f - abs;
    localSupport = fmaxf(last, 0.0f);
    curSupport *= localSupport;

    eval = ptrLevel[m +(2*storageSize)] * locData_2;
    index_calc = eval - ptrIndex[m + (2*storageSize)];
    abs = fabsf(index_calc);
    last = 1.0f - abs;
    localSupport = fmaxf(last, 0.0f);
    curSupport *= localSupport;

    eval = ptrLevel[m + (3*storageSize)] * locData_3;
    index_calc = eval - ptrIndex[m + (3*storageSize)];
    abs = fabsf(index_calc);
    last = 1.0f - abs;
    localSupport = fmaxf(last, 0.0f);
    curSupport *= localSupport;

    eval = ptrLevel[m + (4*storageSize)] * locData_4;
    index_calc = eval - ptrIndex[m + (4*storageSize)];
    abs = fabsf(index_calc);
    last = 1.0f - abs;
    localSupport = fmaxf(last, 0.0f);
    curSupport *= localSupport;

    myResult += curSupport;
  }

  ptrResult[globalIdx] = myResult;
}

double multTransSPCUDA(float* ptrSource, float* ptrGlobalResult, size_t sourceSize, size_t storageSize, size_t dims) {
  // copy coefficients to GPU
  memcpy(host_datavec_sp, ptrSource, sourceSize * sizeof(float));
  hipMemcpy(gpu_datavec_sp, ptrSource, sourceSize * sizeof(float), hipMemcpyHostToDevice);
  int myStorageSize = ((int)storageSize) / 64;

  // Measure time
  sg::base::SGppStopwatch* mytimer = new sg::base::SGppStopwatch();
  mytimer->start();

  if (dims == 5) {
    multTransSP_CUDA_5d <<< myStorageSize, 64>>>(gpu_datavec_sp, gpu_dataset_sp, gpu_grid_level_sp, gpu_grid_index_sp, gpu_alpha_sp, (unsigned int)sourceSize, (unsigned int)gpu_full_storageSize,0);
  } else {
    multTransSP_CUDA <<< myStorageSize, 64>>>(gpu_datavec_sp, gpu_dataset_sp, gpu_grid_level_sp, gpu_grid_index_sp, gpu_alpha_sp, (unsigned int)sourceSize, (unsigned int)gpu_full_storageSize, (unsigned int)dims, 0);
  }

  // copy results back to host
  hipMemcpy(host_alpha_sp, gpu_alpha_sp, storageSize * sizeof(float), hipMemcpyDeviceToHost);
  memcpy(ptrGlobalResult, host_alpha_sp, storageSize * sizeof(float));

  double time = mytimer->stop();
  delete mytimer;

  return time;
}

double multSPCUDA(float* ptrAlpha, float* ptrResult, size_t result_size, size_t storageSize, size_t dims) {
  // copy coefficients to GPU
  memcpy(host_alpha_sp, ptrAlpha, storageSize * sizeof(float));
  hipMemcpy(gpu_alpha_sp, host_alpha_sp, storageSize * sizeof(float), hipMemcpyHostToDevice);

  unsigned int tmp = ((unsigned int)storageSize) / CUDA_BLOCK_SIZE_GPU;
  unsigned int fastStorageSize = tmp * CUDA_BLOCK_SIZE_GPU;

  // Measure time
  sg::base::SGppStopwatch* mytimer = new sg::base::SGppStopwatch();
  mytimer->start();

  if (dims == 5) {
    multSP_CUDA_5d <<< result_size / 64, 64 >>> (gpu_alpha_sp, gpu_dataset_sp, gpu_grid_level_sp, gpu_grid_index_sp, gpu_datavec_sp, fastStorageSize, (unsigned int)storageSize, (unsigned int)result_size, 0);
  } else {
    multSP_CUDA <<< result_size / 64, 64 >>> (gpu_alpha_sp, gpu_dataset_sp, gpu_grid_level_sp, gpu_grid_index_sp, gpu_datavec_sp, fastStorageSize, (unsigned int)storageSize, (unsigned int)dims, (unsigned int)result_size, 0);
  }

  // copy results back to host
  hipMemcpy(host_datavec_sp, gpu_datavec_sp, result_size * sizeof(float), hipMemcpyDeviceToHost);
  memcpy(ptrResult, host_datavec_sp, result_size * sizeof(float));

  double time = mytimer->stop();
  delete mytimer;

  return time;
}

void uploadGridSPCUDA(float* ptrLevel, float* ptrIndex, size_t storageSize, size_t dims) {
  size_t mem_size = storageSize * dims * sizeof(float);
  gpu_full_storageSize = storageSize;

  // allocate memory on GPU
  hipMalloc((void**) &gpu_grid_level_sp, mem_size);
  hipMalloc((void**) &gpu_grid_index_sp, mem_size);
  hipMalloc((void**) &gpu_alpha_sp, storageSize * sizeof(float));
  hipHostMalloc((void**) &host_alpha_sp, storageSize * sizeof(float));
  hipHostMalloc((void**) &host_grid_level_sp, mem_size);
  hipHostMalloc((void**) &host_grid_index_sp, mem_size);

  memset(host_grid_level_sp, 0, mem_size);
  memset(host_grid_index_sp, 0, mem_size);

  // copy and transpose grid level and index to pinned memory
  //#pragma omp parallel for
  for (size_t i = 0; i < storageSize; i++) {
    for (size_t d = 0; d < dims; d++) {
      host_grid_level_sp[(d*storageSize)+i] = ptrLevel[(i*dims)+d];
      host_grid_index_sp[(d*storageSize)+i] = ptrIndex[(i*dims)+d];
    }
  } 

  // copy host memory to device
  hipMemcpy(gpu_grid_level_sp, host_grid_level_sp, mem_size, hipMemcpyHostToDevice);
  hipMemcpy(gpu_grid_index_sp, host_grid_index_sp, mem_size, hipMemcpyHostToDevice);
}

void uploadDataSPCUDA(float* ptrData, size_t dataSize, size_t dims) {
  size_t mem_size = dataSize * dims * sizeof(float);

  // allocate memory on GPU
  hipMalloc((void**) &gpu_dataset_sp, mem_size);
  hipMalloc((void**) &gpu_datavec_sp, dataSize * sizeof(float));
  hipHostMalloc((void**) &host_datavec_sp, dataSize * sizeof(float));

  // copy host memory to device
  hipMemcpy(gpu_dataset_sp, ptrData, mem_size, hipMemcpyHostToDevice);
}

void deleteGridSPCUDA() {
  hipFree(gpu_grid_level_sp);
  hipFree(gpu_grid_index_sp);
  hipFree(gpu_alpha_sp);
  hipHostFree(host_alpha_sp);
  hipHostFree(host_grid_level_sp);
  hipHostFree(host_grid_index_sp);
}

void deleteDataSPCUDA() {
  hipFree(gpu_dataset_sp);
  hipFree(gpu_datavec_sp);
  hipHostFree(host_datavec_sp);
}
