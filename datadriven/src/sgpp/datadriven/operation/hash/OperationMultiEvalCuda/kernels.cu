#include "hip/hip_runtime.h"
// Copyright (C) 2008-today The SG++ project
// This file is part of the SG++ project. For conditions of distribution and
// use, please see the copyright notice provided with SG++ or at
// sgpp.sparsegrids.org

#include <stdint.h>
#include <algorithm>
#include "kernels.cuh"

///@cond DOXY_IGNORE // NOLINT()
namespace sgpp {
namespace datadriven {
namespace OpMultiEvalCudaDetail {

/// Device variables for magic numbers
__constant__ double __idxtable[4];
__constant__ int32_t __idxtablei[4];

__constant__ uint32_t __dDIM[1];

#define __DIM __dDIM[0]

/** Shared gridnode objects. Works up to Dim = 32.
 *  Be aware of limited shared memory on GPU before increasing values!
 */
__shared__ gridnode_t snode[32];

/// Initializes magic numbers and dimension constants on the GPU
void initCudaConstants(const uint32_t DIM) {
  hipMemcpyToSymbol(HIP_SYMBOL(__idxtablei), idxtablei, sizeof(int32_t) * 4);
  hipMemcpyToSymbol(HIP_SYMBOL(__idxtable), idxtable, sizeof(double) * 4);
  hipMemcpyToSymbol(HIP_SYMBOL(__dDIM), &DIM, sizeof(uint32_t));
}

/// Evaluation of a DIM-dim polynomial basis function
/// The node of the basis function is passed by the global shared snode
/// For further code comments see evalBasis_1
__device__ double evalBasis(double*& pos, uint32_t start) {
  double eval = 1.0;
  double x;
  uint32_t id;
  int32_t root;
  for (uint32_t d = 0; d < __DIM; ++d) {
    x = pos[start + d] * snode[d].level2;
    id = x;
    id |= 1;
    if (id != snode[d].index) return 0.0;
    x = static_cast<double>(id + 1) - x;
    eval *= x;
    root = -1;
    x -= 2.0;
    for (int32_t j = 2; j < snode[d].grad; j <<= 1) {
      eval *= x / root;
      root += (__idxtablei[id & 3] * j);
      x += (__idxtablei[id & 3] * j);
      id >>= 1;
    }
    eval *= x / root;
  }
  return eval;
}

/// Evaluation of (DIM-1)-dim polynomial basis function. It skipps the first dimension
/// For further code comments see evalBasis_1
__device__ double evalBasis_d(double* pos, gridnode_t* node) {
  double eval = 1.0;
  double x;
  uint32_t id;
  int32_t root;
  for (uint32_t d = 1; d < __DIM; ++d) {
    x = pos[d] * node[d].level2;
    id = x;
    id |= 1;
    x = static_cast<double>(id + 1) - x;
    eval *= x;
    root = -1;
    x -= 2.0;
    for (int32_t j = 2; j < node[d].grad; j <<= 1) {
      eval *= x / root;
      root += (__idxtablei[id & 3] * j);
      x += (__idxtablei[id & 3] * j);
      id >>= 1;
    }
    eval *= x / root;
  }
  return eval;
}

/// Evaluation of a one-dim polynomial basis function.
__device__ double evalBasis_1(double pos, gridnode_t& node) {
  // This function iterativley constructs the Legandre polynomial
  // The localy nearest ancestors of the node are used as roots
  double eval = 1.0;
  uint32_t id;
  int32_t root;
  // Scale position to level
  pos *= node.level2;
  // Compute index according to the eval position
  id = pos;
  id |= 1;
  // Go to right support boundary as 1st root
  pos = static_cast<double>(id + 1) - pos;
  eval *= pos;
  root = -1;
  // Go to left support boundary for 2nd root
  pos -= 2.0;
  for (int32_t j = 2; j < node.grad; j <<= 1) {
    // Multiply new root
    eval *= pos / root;
    // Use magic numbers and shifted level to find next ancestor
    root += (__idxtablei[id & 3] * j);
    pos += (__idxtablei[id & 3] * j);
    // Shift the level
    id >>= 1;
  }
  // Multiply last root
  eval *= pos / root;
  return eval;
}

__global__ void gpu_zindex(double* pos, uint64_t* index) {
  uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  pos = &pos[idx * __DIM];
  uint64_t val = 0;
  // uint32_t bits = 64/__DIM;
  for (uint32_t i = 0; i < 64 / __DIM; ++i) {
    for (uint32_t d = 0; d < __DIM; ++d) {
      val <<= 1;
      if (pos[d] >= 1.0)
        val |= 1;
      else
        val |= static_cast<uint64_t>(pos[d] * (1 << i)) & 1;
    }
  }
  index[idx] = val;
  // index[idx] = zvalue(&pos[idx*__DIM]);
}

/// Standard evaluation. Each parallel thread computes one evaluation.
__global__ void gpu_eval(double* res, gridnode_t* node, double* a, double* pos, uint32_t* limit,
                         uint32_t subcnt, uint32_t* subs) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  res = &res[idx];
  pos = &pos[idx * __DIM];
  limit = &limit[idx * __DIM];

  double tmp, poly;
  double data = 0.0;
  uint32_t path;

  // Iterate through all possible subspaces with levelsum = n + d - 1
  // The subspaces are pre-computed
  for (uint32_t j = 0; j < subcnt * __DIM; j += __DIM) {
    // Start with root node
    idx = GRID_START;

    // Follow path in all but thre first dimension
    for (uint32_t d = 1; d < __DIM; ++d) {
      // Skip if evaluation point is outside of any support
      if (subs[j + d] > limit[d]) {
        idx = GRID_END;
        break;
      }
      // Follow path to children
      path = pos[d] * (1 << subs[j + d]);
      path = __brev(path) >> (32 - subs[j + d]);
      for (uint32_t l = 1; l < subs[j + d]; ++l) {
        if (idx == GRID_END) break;
        idx = node[idx * __DIM + d].child[path & 1];
        path >>= 1;
      }
      // Skip if leaf is reached
      if (idx == GRID_END) break;
    }
    if (idx == GRID_END) continue;

    // Do the actual evaluation along the remaining one-dim subtree
    path = pos[0] * (1 << subs[j]);
    path = __brev(path) >> (32 - subs[j]);
    tmp = 0.0;
    poly = evalBasis_d(pos, &node[idx * __DIM]);
    // Go down tree and accumulate weighted basis functions
    for (uint32_t l = 1; l <= subs[j]; ++l) {
      if (idx == GRID_END) break;
      tmp += a[idx] * evalBasis_1(pos[0], node[idx * __DIM]);
      idx = node[idx * __DIM].child[(path & 1)];
      path >>= 1;
    }
    // Geather for all possible subspaces
    data += tmp * poly;
  }
  res[0] = data;
}

/** Transposed evaluation with optimized streaming apporach and additional FMA.
 *  Each thread block computes one result entry
 */
__global__ void gpu_transevel(double* a, gridnode_t* node, double* pos, double* y, limit_t* limit,
                              double* b, double c, double M_) {
  __shared__ double val[CUDA_BLOCKSIZE];
  __shared__ limit_t slimit;
  double tmp = 0.0;
  if (threadIdx.x < __DIM) {
    snode[threadIdx.x] = node[blockIdx.x * __DIM + threadIdx.x];
  }
  slimit = limit[blockIdx.x];
  __syncthreads();

  // Streaming loop strided within the thread block
  // The limits can be pre-computed if the dataset is aligned along a space-filling curve
  // Otherwise use 0 and M as limits
  for (uint32_t i = slimit.lower + threadIdx.x; i < slimit.upper; i += CUDA_BLOCKSIZE) {
    tmp += y[i] * evalBasis(pos, i * __DIM);
  }
  val[threadIdx.x] = tmp;

  // Fan-in all sub results
  __syncthreads();
#if CUDA_BLOCKSIZE > 512
  if (threadIdx.x < 512) val[threadIdx.x] += val[threadIdx.x + 512];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 256
  if (threadIdx.x < 256) val[threadIdx.x] += val[threadIdx.x + 256];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 128
  if (threadIdx.x < 128) val[threadIdx.x] += val[threadIdx.x + 128];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 64
  if (threadIdx.x < 64) val[threadIdx.x] += val[threadIdx.x + 64];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 32
  if (threadIdx.x < 32) val[threadIdx.x] += val[threadIdx.x + 32];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 1];
  __syncthreads();

  // Compute final result
  if (threadIdx.x == 0) {
    tmp = val[0] * M_;
    a[blockIdx.x] = (c * b[blockIdx.x]) + tmp;
  }
}

/// Compute subspace limits for each evaluation point
__global__ void gpu_preprocess(uint32_t* limit, gridnode_t* node, double* pos, uint32_t maxlevel) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  pos = &pos[idx * __DIM];
  limit = &limit[idx * __DIM];
  uint32_t path;
  for (uint32_t d = 0; d < __DIM; ++d) {
    idx = GRID_START;
    path = pos[d] * (1 << maxlevel);
    path = __brev(path) >> (32 - maxlevel);
    limit[d] = maxlevel;
    for (uint32_t l = 1; l <= maxlevel; ++l) {
      if (idx == GRID_END) {
        limit[d] = l;
        break;
      }
      idx = node[idx * __DIM + d].child[path & 1];
      path >>= 1;
    }
  }
}

/// Transposed evaluation with optimized streaming approach
/// See the other gpu_transeval kernel for code comments
__global__ void gpu_transevel(double* a, gridnode_t* node, double* pos, double* y, limit_t* limit) {
  __shared__ double val[CUDA_BLOCKSIZE];
  // __shared__ gridnode_t snode[32];
  double tmp = 0.0;
  if (threadIdx.x < __DIM) {
    snode[threadIdx.x] = node[blockIdx.x * __DIM + threadIdx.x];
  }
  __syncthreads();
  for (uint32_t i = limit[blockIdx.x].lower + threadIdx.x; i < limit[blockIdx.x].upper;
       i += CUDA_BLOCKSIZE) {
    // tmp += y[i] * evalBasis(&pos[i*__DIM], snode);//&node[blockIdx.x*__DIM]);
    tmp += y[i] * evalBasis(pos, i * __DIM);
  }
  val[threadIdx.x] = tmp;
  __syncthreads();
#if CUDA_BLOCKSIZE > 512
  if (threadIdx.x < 512) val[threadIdx.x] += val[threadIdx.x + 512];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 256
  if (threadIdx.x < 256) val[threadIdx.x] += val[threadIdx.x + 256];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 128
  if (threadIdx.x < 128) val[threadIdx.x] += val[threadIdx.x + 128];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 64
  if (threadIdx.x < 64) val[threadIdx.x] += val[threadIdx.x + 64];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
#if CUDA_BLOCKSIZE > 32
  if (threadIdx.x < 32) val[threadIdx.x] += val[threadIdx.x + 32];
  __syncthreads();
#endif  // CUDA_BLOCKSIZE
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x < 16) val[threadIdx.x] += val[threadIdx.x + 1];
  __syncthreads();
  if (threadIdx.x == 0) a[blockIdx.x] = val[threadIdx.x];
}

/// Compute streaming boundary limits for datasets aligned along a Morton order curve
__global__ void gpu_zbound(uint64_t* idx_p, gridnode_t* node, limit_t* limit, uint32_t M) {
  // Use the orderd dataset as binary search tree to find position of upper and lower
  // support corner within the space-filling curve. Each thread computes the limits for
  // one grid node.
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  node = &node[idx * __DIM];
  uint64_t idx_l, idx_h;
  double xl, xh;
  idx_h = 0;
  idx_l = 0;
  uint32_t bits = 64 / __DIM;
  for (uint32_t i = 0; i < bits; ++i) {
    for (uint32_t d = 0; d < __DIM; ++d) {
      xl = node[d].x - 1.0 / node[d].level2;
      xh = node[d].x + 0.8 / node[d].level2;
      idx_h <<= 1;
      idx_h |= uint64_t(xh * (1 << i)) & 1;
      idx_l <<= 1;
      idx_l |= uint64_t(xl * (1 << i)) & 1;
    }
  }
  uint32_t l, h;
  l = 0;
  h = 0;
  for (uint32_t i = M / 2; i > 0; i >>= 1) {
    if (idx_p[i + l] < idx_l) l += i;
    if (idx_p[i + h] <= idx_h) h += i;
  }
  limit[idx].lower = l & (0xFFFFFFFF - (CUDA_BLOCKSIZE - 1));
  limit[idx].upper = min((h + 2) | (CUDA_BLOCKSIZE - 1), M);
}

}  // namespace OpMultiEvalCudaDetail
}  // namespace datadriven
}  // namespace sgpp
///@endcond // NOLINT()
