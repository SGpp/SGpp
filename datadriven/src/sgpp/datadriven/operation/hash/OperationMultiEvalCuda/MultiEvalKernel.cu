// Copyright (C) 2008-today The SG++ project
// This file is part of the SG++ project. For conditions of distribution and
// use, please see the copyright notice provided with SG++ or at
// sgpp.sparsegrids.org

#include "MultiEvalKernel.hpp"
#include "basicCuda.hpp"
#include "consts.hpp"
#include "cudaHelper.hpp"
#include "kernels.cuh"

///@cond DOXY_IGNORE // NOLINT()
namespace sgpp {
namespace datadriven {
namespace OpMultiEvalCudaDetail {

/// Wrapper for kernel call of the stream boundary limitation
void streamboundCuda(double* pos, gridnode_t* node, limit_t* limit, uint32_t M, uint32_t _M,
                     uint32_t N) {
  uint64_t* idx_p;
  hipMalloc(reinterpret_cast<void**>(&idx_p), sizeof(uint64_t) * M);
  hipDeviceSynchronize();
  CudaCheckError();
  // Compute index of the Morton order curve
  gpu_zindex<<<M / CUDA_BLOCKSIZE, CUDA_BLOCKSIZE>>>(pos, idx_p);
  hipDeviceSynchronize();
  CudaCheckError();
  // Compute lower and upper boundaries
  gpu_zbound<<<N / CUDA_BLOCKSIZE, CUDA_BLOCKSIZE>>>(idx_p, node, limit, _M);
  hipDeviceSynchronize();
  CudaCheckError();
  hipFree(idx_p);
}

/// Wrapper for kernel call of subspace limits for standard eval
void preprocessCuda(gridnode_t* node, double* pos, uint32_t* limit, uint32_t maxlevel, uint32_t M,
                    uint32_t DIM) {
  initCudaConstants(DIM);
  gpu_preprocess<<<(M / CUDA_BLOCKSIZE), CUDA_BLOCKSIZE>>>(limit, node, pos, maxlevel);
  hipDeviceSynchronize();
}

/// Wrapper for kernel call of standard evaluation
void evalCuda(double* res, double* a, gridnode_t* node, double* pos, uint32_t M, uint32_t maxlevel,
              uint32_t* limit, uint32_t subcnt, uint32_t* subs) {
  gpu_eval<<<(M / CUDA_BLOCKSIZE), CUDA_BLOCKSIZE>>>(res, node, a, pos, limit, subcnt, subs);
  hipDeviceSynchronize();
}

/// Wrapper for kernel call of transposed eval with additional FMA
void transposedCuda(double* a, gridnode_t* node, double* pos, double* y, limit_t* limit, double* b,
                    double c, uint32_t M, uint32_t _M, uint32_t N) {
  gpu_transevel<<<N, CUDA_BLOCKSIZE>>>(a, node, pos, y, limit, b, c, 1.0 / static_cast<double>(_M));
  hipDeviceSynchronize();
}

/// Wrapper for kernel call of transposed eval
void transposedCuda(double* a, gridnode_t* node, double* pos, double* y, limit_t* limit,
                    uint32_t N) {
  gpu_transevel<<<N, CUDA_BLOCKSIZE>>>(a, node, pos, y, limit);
  hipDeviceSynchronize();
}

}  // namespace OpMultiEvalCudaDetail
}  // namespace datadriven
}  // namespace sgpp
///@endcon // NOLINT()
