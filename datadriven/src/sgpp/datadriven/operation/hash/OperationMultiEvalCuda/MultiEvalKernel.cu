#include "basicCuda.hpp"
#include "cudaHelper.hpp"
#include "kernels.cuh"
#include "MultiEvalKernel.hpp"
#include "consts.hpp"

/// Wrapper for kernel call of the stream boundary limitation
void streamboundCuda (double* pos, gridnode_t* node, limit_t* limit, uint32_t M, uint32_t N) {
  uint64_t *idx_p;
	hipMalloc((void**)&idx_p,sizeof(uint64_t)*M);
  hipDeviceSynchronize();
  CudaCheckError();
  // Compute index of the Morton order curve
  gpu_zindex<<<M/CUDA_BLOCKSIZE,CUDA_BLOCKSIZE>>>(pos, idx_p);
 	hipDeviceSynchronize();
  CudaCheckError();
  // Compute lower and upper boundaries
  gpu_zbound<<<N/CUDA_BLOCKSIZE,CUDA_BLOCKSIZE>>>(idx_p, node, limit, M);
 	hipDeviceSynchronize();
  CudaCheckError();
  hipFree(idx_p);
}

/// Wrapper for kernel call of subspace limits for standard eval
void preprocessCuda (gridnode_t* node, double* pos, uint32_t* limit, uint32_t maxlevel, uint32_t M,
    uint32_t DIM) {
	initCudaConstants(DIM);
  gpu_preprocess<<<(M/CUDA_BLOCKSIZE),CUDA_BLOCKSIZE>>>(limit, node, pos, maxlevel);
  hipDeviceSynchronize();
}

/// Wrapper for kernel call of standard evaluation
void evalCuda (double* res, double *a, gridnode_t* node, double* pos, uint32_t M,
  uint32_t maxlevel, uint32_t* limit, uint32_t subcnt, uint32_t* subs) {
  gpu_eval<<<(M/CUDA_BLOCKSIZE),CUDA_BLOCKSIZE>>>(res, node, a, pos, limit, subcnt, subs);
  hipDeviceSynchronize();
}

/// Wrapper for kernel call of transposed eval with additional FMA
void transposedCuda (double* a, gridnode_t* node, double* pos, double* y,
    limit_t* limit, double* b, double c, uint32_t M, uint32_t N) {
  gpu_transevel<<<N,CUDA_BLOCKSIZE>>>(a, node, pos, y, limit, b, c, 1.0/double(M));
  hipDeviceSynchronize();
}

/// Wrapper for kernel call of transposed eval
void transposedCuda (double* a, gridnode_t* node, double* pos, double* y,
    limit_t* limit, uint32_t M, uint32_t N) {
  gpu_transevel<<<N,CUDA_BLOCKSIZE>>>(a, node, pos, y, limit);
  hipDeviceSynchronize();
}
